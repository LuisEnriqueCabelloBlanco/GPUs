#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <png_io.h>

#include "routinesGPU.h"

void canny(uint8_t *im, uint8_t *image_out,
	float *NR, float *G, float *phi, float *Gx, float *Gy, uint8_t *pedge,
	float level,
	int height, int width){

		noise_reduc<<<1,1>>>(im,NR,height,width);
		gradient_img<<<1,1>>>(NR,G,phi,Gx,Gy,height,width);
		edge<<<1,1>>>();
		hyteresis_Thresholding<<<1,1>>>();


}


__global__ void noise_reduc(uint8_t*im, float*NR ,int height,int width){

	int i, j;
	for(i=2; i<height-2; i++)
		for(j=2; j<width-2; j++)
		{
			// Noise reduction
			NR[i*width+j] =
				 (2.0*im[(i-2)*width+(j-2)] +  4.0*im[(i-2)*width+(j-1)] +  5.0*im[(i-2)*width+(j)] +  4.0*im[(i-2)*width+(j+1)] + 2.0*im[(i-2)*width+(j+2)]
				+ 4.0*im[(i-1)*width+(j-2)] +  9.0*im[(i-1)*width+(j-1)] + 12.0*im[(i-1)*width+(j)] +  9.0*im[(i-1)*width+(j+1)] + 4.0*im[(i-1)*width+(j+2)]
				+ 5.0*im[(i  )*width+(j-2)] + 12.0*im[(i  )*width+(j-1)] + 15.0*im[(i  )*width+(j)] + 12.0*im[(i  )*width+(j+1)] + 5.0*im[(i  )*width+(j+2)]
				+ 4.0*im[(i+1)*width+(j-2)] +  9.0*im[(i+1)*width+(j-1)] + 12.0*im[(i+1)*width+(j)] +  9.0*im[(i+1)*width+(j+1)] + 4.0*im[(i+1)*width+(j+2)]
				+ 2.0*im[(i+2)*width+(j-2)] +  4.0*im[(i+2)*width+(j-1)] +  5.0*im[(i+2)*width+(j)] +  4.0*im[(i+2)*width+(j+1)] + 2.0*im[(i+2)*width+(j+2)])
				/159.0;
		}
}

__global__ void gradient_img(float *NR,float *G, float *phi, float *Gx, float *Gy,int height,int width){
	int i, j;

	float PI = 3.141593;

	for(i=2; i<height-2; i++)
		for(j=2; j<width-2; j++)
		{
			// Intensity gradient of the image
			Gx[i*width+j] = 
				 (1.0*NR[(i-2)*width+(j-2)] +  2.0*NR[(i-2)*width+(j-1)] +  (-2.0)*NR[(i-2)*width+(j+1)] + (-1.0)*NR[(i-2)*width+(j+2)]
				+ 4.0*NR[(i-1)*width+(j-2)] +  8.0*NR[(i-1)*width+(j-1)] +  (-8.0)*NR[(i-1)*width+(j+1)] + (-4.0)*NR[(i-1)*width+(j+2)]
				+ 6.0*NR[(i  )*width+(j-2)] + 12.0*NR[(i  )*width+(j-1)] + (-12.0)*NR[(i  )*width+(j+1)] + (-6.0)*NR[(i  )*width+(j+2)]
				+ 4.0*NR[(i+1)*width+(j-2)] +  8.0*NR[(i+1)*width+(j-1)] +  (-8.0)*NR[(i+1)*width+(j+1)] + (-4.0)*NR[(i+1)*width+(j+2)]
				+ 1.0*NR[(i+2)*width+(j-2)] +  2.0*NR[(i+2)*width+(j-1)] +  (-2.0)*NR[(i+2)*width+(j+1)] + (-1.0)*NR[(i+2)*width+(j+2)]);


			Gy[i*width+j] = 
				 ((-1.0)*NR[(i-2)*width+(j-2)] + (-4.0)*NR[(i-2)*width+(j-1)] +  (-6.0)*NR[(i-2)*width+(j)] + (-4.0)*NR[(i-2)*width+(j+1)] + (-1.0)*NR[(i-2)*width+(j+2)]
				+ (-2.0)*NR[(i-1)*width+(j-2)] + (-8.0)*NR[(i-1)*width+(j-1)] + (-12.0)*NR[(i-1)*width+(j)] + (-8.0)*NR[(i-1)*width+(j+1)] + (-2.0)*NR[(i-1)*width+(j+2)]
				+    2.0*NR[(i+1)*width+(j-2)] +    8.0*NR[(i+1)*width+(j-1)] +    12.0*NR[(i+1)*width+(j)] +    8.0*NR[(i+1)*width+(j+1)] +    2.0*NR[(i+1)*width+(j+2)]
				+    1.0*NR[(i+2)*width+(j-2)] +    4.0*NR[(i+2)*width+(j-1)] +     6.0*NR[(i+2)*width+(j)] +    4.0*NR[(i+2)*width+(j+1)] +    1.0*NR[(i+2)*width+(j+2)]);

			G[i*width+j]   = sqrtf((Gx[i*width+j]*Gx[i*width+j])+(Gy[i*width+j]*Gy[i*width+j]));	//G = √Gx²+Gy²
			phi[i*width+j] = atan2f(fabs(Gy[i*width+j]),fabs(Gx[i*width+j]));

			if(fabs(phi[i*width+j])<=PI/8 )
				phi[i*width+j] = 0;
			else if (fabs(phi[i*width+j])<= 3*(PI/8))
				phi[i*width+j] = 45;
			else if (fabs(phi[i*width+j]) <= 5*(PI/8))
				phi[i*width+j] = 90;
			else if (fabs(phi[i*width+j]) <= 7*(PI/8))
				phi[i*width+j] = 135;
			else phi[i*width+j] = 0;
	}
}

void lane_assist_GPU(uint8_t *im, int height, int width,
	int *x1, int *y1, int *x2, int *y2, int *nlines)
{
	// Create temporal buffers 
	uint8_t *imEdge = (uint8_t *)malloc(sizeof(uint8_t) * width * height);
	float *NR = (float *)malloc(sizeof(float) * width * height);
	float *G = (float *)malloc(sizeof(float) * width * height);
	float *phi = (float *)malloc(sizeof(float) * width * height);
	float *Gx = (float *)malloc(sizeof(float) * width * height);
	float *Gy = (float *)malloc(sizeof(float) * width * height);
	uint8_t *pedge = (uint8_t *)malloc(sizeof(uint8_t) * width * height);

	/* Canny */
	canny(im, imEdge,
		NR, G, phi, Gx, Gy, pedge,
		1000.0f, //level
		height, width);

	write_png_fileBW("out_edges.png",imEdge,width,height);
	/* To do */
}
